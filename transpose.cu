#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

/* Time */
#include <sys/time.h>
#include <sys/resource.h>

static struct timeval tv0;
double getMicroSeconds()
{
	double t;
	gettimeofday(&tv0, (struct timezone*)0);
	t = ((tv0.tv_usec) + (tv0.tv_sec)*1000000);

	return (t);
}

void init_seed()
{
	int seedi=1;
	FILE *fd;

	/* Generated random values between 0.00 - 1.00 */
	fd = fopen("/dev/urandom", "r");
	fread( &seedi, sizeof(int), 1, fd);
	fclose (fd);
	srand( seedi );
}

void init2Drand(float **buffer, int n)
{
	int i, j;

	for (i=0; i<n; i++)
		for(j=0; j<n; j++)
			buffer[i][j] = 500.0*(float(rand())/RAND_MAX)-500.0; /* [-500 500]*/
}

float *getmemory1D( int nx )
{
	int i;
	float *buffer;

	if( (buffer=(float *)malloc(nx*sizeof(float *)))== NULL )
	{
		fprintf( stderr, "ERROR in memory allocation\n" );
		return( NULL );
	}

	for( i=0; i<nx; i++ )
		buffer[i] = 0.0;

	return( buffer );
}


float **getmemory2D(int nx, int ny)
{
	int i,j;
	float **buffer;

	if( (buffer=(float **)malloc(nx*sizeof(float *)))== NULL )
	{
		fprintf( stderr, "ERROR in memory allocation\n" );
		return( NULL );
	}

	if( (buffer[0]=(float *)malloc(nx*ny*sizeof(float)))==NULL )
	{
		fprintf( stderr, "ERROR in memory allocation\n" );
		free( buffer );
		return( NULL );
	}

	for( i=1; i<nx; i++ )
	{
		buffer[i] = buffer[i-1] + ny;
	}

	for( i=0; i<nx; i++ )
		for( j=0; j<ny; j++ )
		{
			buffer[i][j] = 0.0;
		}

	return( buffer );
}



/********************************************************************************/
/********************************************************************************/

/*
 * Traspose 2D version
 */
void transpose2D(float **in, float **out, int n)
{
	int i, j;

	for(j=0; j < n; j++) 
		for(i=0; i < n; i++) 
			out[j][i] = in[i][j]; 
}

/*
 * Traspose 1D version
 */
void transpose1D(float *in, float *out, int n)
{
	int i, j;

	for(j=0; j < n; j++) 
		for(i=0; i < n; i++) 
			out[j*n+i] = in[i*n+j]; 
}

/*
 * Traspose CUDA version
 */

#define NTHREADS1D 256

__global__ void transpose_device(float *in, float *out, int rows, int cols) 
{ 
	int i, j; 
	i = blockIdx.x * blockDim.x + threadIdx.x; 

	if (i<rows)
		for ( j=0; j<cols; j++) 
			out [ i * rows + j ] = in [ j * cols + i ]; 
}

int check(float *GPU, float *CPU, int n)
{
	int i;

	for (i=0; i<n; i++)
		if(GPU[i]!=CPU[i])
			return(1);

	return(0);
}

int main(int argc, char **argv)
{
	int n;
	float **array2D, **array2D_trans;
	float *array1D,  *array1D_trans;
	float *array1D_trans_GPU;

	double t0, t1;

	if (argc==2)
		n = atoi(argv[1]);
	else {
		n = 8192;
		printf("./exec n (by default n=%i)\n", n);
	}
	
	/* Initizalization */
	init_seed();
	array2D       = getmemory2D(n,n);
	array2D_trans = getmemory2D(n,n);
	array1D_trans_GPU = getmemory1D(n*n);
	array1D       = array2D[0];
	array1D_trans = array2D_trans[0];
	init2Drand(array2D, n);

	/* Transpose 1D version */
	t0 = getMicroSeconds();
	transpose1D(array1D, array1D_trans, n);
	printf("Transpose version 1D: %f MB/s\n", n*n*sizeof(float)/((getMicroSeconds()-t0)/1000000)/1024/1024);


	/* CUDA vesion */
	float *darray1D, *darray1D_trans;
	hipMalloc((void**)&darray1D, n*n*sizeof(float));
	hipMemcpy(darray1D, array1D, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&darray1D_trans, n*n*sizeof(float));

	dim3 dimBlock(NTHREADS1D);
	int blocks = n/NTHREADS1D;
		if (n%NTHREADS1D>0) blocks++;
	dim3 dimGrid(blocks);

	t0 = getMicroSeconds();
	transpose_device<<<dimGrid,dimBlock>>>(darray1D, darray1D_trans, n, n);	
	hipDeviceSynchronize();
	t1 = getMicroSeconds();
	hipMemcpy(array1D_trans_GPU, darray1D_trans, n*n*sizeof(float), hipMemcpyDeviceToHost);

	
	printf("Transpose kernel version: %f MB/s tKernel=%f (us)\n", n*n*sizeof(float)/((getMicroSeconds()-t0)/1000000)/1024/1024, (t1-t0)/1000000);
	

	if (check(array1D_trans_GPU, array1D_trans, n*n))
		printf("Transpose CPU-GPU differs!!\n");


	return(0);
}
